/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/
#ifndef GENERAL_PARSER_GPU_CU_
#define GENERAL_PARSER_GPU_CU_
#include "general_parser_gpu.h"

using namespace hesai::lidar;
template <typename T_Point>
GeneralParserGpu<T_Point>::GeneralParserGpu(uint16_t maxPacket, uint16_t maxPoint) {
  cudaSafeMalloc(correction_azi_cu_, sizeof(double) * DEFAULT_MAX_LASER_NUM);
  cudaSafeMalloc(correction_ele_cu_, sizeof(double) * DEFAULT_MAX_LASER_NUM);
  cudaSafeMalloc(firetimes_cu_, sizeof(double) * DEFAULT_MAX_LASER_NUM);
  if (maxPacket > 0 && maxPoint > 0) {
    cudaSafeMalloc(point_data_cu_, sizeof(PointDecodeData) * maxPacket * maxPoint);
    cudaSafeMalloc(packet_data_cu_, sizeof(PacketDecodeData) * maxPacket);
    cudaSafeMalloc(points_cu_, sizeof(LidarPointXYZDAE) * maxPacket * maxPoint);
    cudaSafeMalloc(valid_points_cu_, sizeof(uint32_t) * maxPacket);
    points_ = new LidarPointXYZDAE[maxPacket * maxPoint];
  } else {
    point_data_cu_ = nullptr;
    packet_data_cu_ = nullptr;
    points_cu_ = nullptr;
    valid_points_cu_ = nullptr;
    points_ = nullptr;
  }
}
template <typename T_Point>
GeneralParserGpu<T_Point>::~GeneralParserGpu() {
  cudaSafeFree(correction_azi_cu_);
  cudaSafeFree(correction_ele_cu_);
  cudaSafeFree(firetimes_cu_);
  if (point_data_cu_ != nullptr) cudaSafeFree(point_data_cu_);
  if (packet_data_cu_ != nullptr) cudaSafeFree(packet_data_cu_);
  if (points_cu_ != nullptr) cudaSafeFree(points_cu_);
  if (valid_points_cu_ != nullptr) cudaSafeFree(valid_points_cu_);
  if (points_ != nullptr) delete[] points_;
  if (point_cloud_cu_ != nullptr) cudaSafeFree(point_cloud_cu_);
}

template <typename T_Point>
void GeneralParserGpu<T_Point>::LoadCorrectionStruct(void* _correction) {
  correction_ptr = (CorrectionData*)_correction;
  CUDACheck(hipMemcpy(correction_azi_cu_, correction_ptr->azimuth, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
  CUDACheck(hipMemcpy(correction_ele_cu_, correction_ptr->elevation, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
}

template <typename T_Point>
void GeneralParserGpu<T_Point>::LoadFiretimesStruct(void* _firetimes) {
  firetimes_ptr = (float*)_firetimes;
  CUDACheck(hipMemcpy(firetimes_cu_, firetimes_ptr, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
}

template <typename T_Point>
void GeneralParserGpu<T_Point>::updateCorrectionFile() {
  if (*get_correction_file_ && correction_load_sequence_num_cuda_use_ != *correction_load_sequence_num_) {
    correction_load_sequence_num_cuda_use_ = *correction_load_sequence_num_;
    CUDACheck(hipMemcpy(correction_azi_cu_, correction_ptr->azimuth, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
    CUDACheck(hipMemcpy(correction_ele_cu_, correction_ptr->elevation, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
  }
}

template <typename T_Point>
void GeneralParserGpu<T_Point>::updateFiretimeFile() {
  if (*get_firetime_file_ && firetime_load_sequence_num_cuda_use_ != *firetime_load_sequence_num_) {
    firetime_load_sequence_num_cuda_use_ = *firetime_load_sequence_num_;
    CUDACheck(hipMemcpy(firetimes_cu_, firetimes_ptr, sizeof(double) * DEFAULT_MAX_LASER_NUM, hipMemcpyHostToDevice));
  }
}

template <typename T_Point>
void GeneralParserGpu<T_Point>::DoRemake(float azi_, float elev_, const RemakeConfig &remake_config, int &point_idx) {
  if (remake_config.flag == false) return;
  point_idx = -1;
  elev_ = elev_ > 180.0 ? elev_ - 360.0 : elev_;
  int new_azi_iscan = (azi_ - remake_config.min_azi) / remake_config.ring_azi_resolution;
  int new_elev_iscan = (elev_ - remake_config.min_elev) / remake_config.ring_elev_resolution;
  if (new_azi_iscan >= 0 && new_azi_iscan < remake_config.max_azi_scan && new_elev_iscan >= 0 && new_elev_iscan < remake_config.max_elev_scan) {
    point_idx = new_azi_iscan * remake_config.max_elev_scan + new_elev_iscan;
  }
}


#endif //GENERAL_PARSER_GPU_CU_