/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/
#ifndef Udp_P40_PARSER_GPU_CU_
#define Udp_P40_PARSER_GPU_CU_
#include "udp_p40_parser_gpu.h"

using namespace hesai::lidar;
template <typename T_Point>
UdpP40ParserGpu<T_Point>::UdpP40ParserGpu(uint16_t maxPacket, uint16_t maxPoint) : GeneralParserGpu<T_Point>(maxPacket, maxPoint) {
  this->optical_center.setNoFlag(LidarOpticalCenter{-0.012, 0.03873, 0});
}
template <typename T_Point>
UdpP40ParserGpu<T_Point>::~UdpP40ParserGpu() {
}

template <typename T_Point>
int UdpP40ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!*this->get_correction_file_) return int(ReturnCode::CorrectionsUnloaded);       
  cudaSafeCall(hipMemcpy(this->point_data_cu_, frame.pointData,
                          frame.per_points_num * frame.packet_num * sizeof(PointDecodeData), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError);
  cudaSafeCall(hipMemcpy(this->packet_data_cu_, frame.packetData,
                          frame.packet_num * sizeof(PacketDecodeData), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError); 
  cudaSafeCall(hipMemcpy(this->valid_points_cu_, frame.valid_points,
                          frame.packet_num * sizeof(uint32_t), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError); 
  this->updateCorrectionFile();
  this->updateFiretimeFile();
  FrameDecodeParam cuda_Param = frame.fParam;
  cuda_Param.firetimes_flag = *this->get_firetime_file_ ? cuda_Param.firetimes_flag : false;
  int ret = compute_p40_cuda(this->points_cu_, this->correction_azi_cu_, this->correction_ele_cu_, this->firetimes_cu_, 
    this->point_data_cu_, this->packet_data_cu_, this->valid_points_cu_, frame.distance_unit, this->optical_center, cuda_Param,
    frame.packet_num, frame.per_points_num);
  if (ret != 0) return ret;

  cudaSafeCall(hipMemcpy(this->points_, this->points_cu_,
                          frame.per_points_num * frame.packet_num * sizeof(LidarPointXYZDAE), 
                          hipMemcpyDeviceToHost), ReturnCode::CudaMemcpyDeviceToHostError);
  for (uint32_t i = 0; i < frame.packet_num; i++) {
    uint32_t point_index = i * frame.per_points_num;
    int point_num = 0;
    for (uint32_t j = point_index; j < point_index + frame.valid_points[i]; j++) {
      if (frame.fParam.config.fov_start != -1 && frame.fParam.config.fov_end != -1) {
        int fov_transfer = this->points_[j].azimuthCalib / M_PI * HALF_CIRCLE;
        if (fov_transfer < frame.fParam.config.fov_start || fov_transfer > frame.fParam.config.fov_end) { //不在fov范围continue
          continue;
        }
      }
      PUT_POINT_IN_POINT_INFO
        uint64_t timestamp = packetData.t.sensor_timestamp * kMicrosecondToNanosecondInt + pointData.data.dnsOff.ns_offset;
        set_timestamp(ptinfo, double(packetData.t.sensor_timestamp) / kMicrosecondToSecond);
        set_timeSecond(ptinfo, timestamp / kNanosecondToSecondInt);
        set_timeNanosecond(ptinfo, timestamp % kNanosecondToSecondInt);

        point_num++;
      }
    }
    frame.valid_points[i] = point_num;
  }
  return 0;
}

#endif //Udp_P40_PARSER_GPU_CU_